#include "hip/hip_runtime.h"
/* 
dan Sandin 8-22-10
*/

#include "CudaParticle.h"

__constant__ float  refldata[REFL_DATA_MUNB][REFL_DATA_ROWS][REFL_DATA_ROW_ELEM];
__constant__ float  injdata[INJT_DATA_MUNB][INJT_DATA_ROWS][INJT_DATA_ROW_ELEM];

void setReflData(void * data, int size)
{
    hipMemcpyToSymbol(HIP_SYMBOL(refldata),data,size);
}

void setInjData(void * data, int size)
{
    hipMemcpyToSymbol(HIP_SYMBOL(injdata),data,size);
}

void launchPoint1(float3* pos, float4* color, float * pdata,float * debugData ,unsigned int width,
    unsigned int height, int max_age,int disappear_age,float alphaControl, float time, float gravity, float colorFreq, float r3)
{
    dim3 block(8,8,1);
    dim3 grid(CUDA_MESH_WIDTH / 8, CUDA_MESH_HEIGHT / 8, 1);
    Point1<<< grid, block>>>(pos,color,pdata,debugData,width,height,max_age,disappear_age,alphaControl,time,gravity,colorFreq,r3);
}

///////////////////////////////////////////////////////////////////////////////
//! Simple partical system
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////

__device__ void  injector0(unsigned int arrayLoc,unsigned int posLoc,float time,float4* pos, float* pdata){
    //ovels
    //  sin(time) + x index/width, x  y are randomly selected because of randon nature of age
    // x afects angular velocity distribution x,y afects liniar velocity distribution
    //     pdata[arrayLoc+1] = 0.02 * (sin(time/5 + (float)x/(float)width/10.0) * (float)(x * y )/ (float)(width * height)/1.0f ) ;//x velocity  sin(time) + x index/width, x is randomly selected because of randon nature of age
    pdata[arrayLoc+1] = 0.02 * (sin(time/5  + (pdata[arrayLoc+5] + 1)/50) * ( ((pdata[arrayLoc+5]  +1)/1 ) * (pdata[arrayLoc+4] + 1.0)/1)  ) ;//x velocity  sin(time) + x index/width, x is randomly selected because of randon nature of age

    pdata[arrayLoc+2] = 0;
    //ovels       
    //       	pdata[arrayLoc+3] = 0.02 * (cos(time/5 + (float)x/(float)width/10.0) * (float)(x *y) / (float)(width * height)/1.0f );// y velocity
    pdata[arrayLoc+3] = 0.02 * (cos(time/5  + (pdata[arrayLoc+5] + 1)/50)  *( ((pdata[arrayLoc+5]  +1)/1 ) * (pdata[arrayLoc+4] + 1.0)/1));// y velocity


    // maybe move the generation point around?


    {
	pos[posLoc].x = 0;
	pos[posLoc].y = 0.5;
	pos[posLoc].z = 0;
    }
}
__device__ float  distRnd1( float seed, int iter){

    unsigned int rndint1;
    rndint1 = (unsigned int)(((seed +1.0)/2.0) *32768) % 32768;
    float sum ;
    sum =0;
    for ( int i = 0;i<iter;i++)
    {
	rndint1 = ((rndint1 * 1103515245 + 12345)/65536) % 32768;
	sum = sum +  0.0002 * (rndint1 % 10000) -1.0;
    }

    return	sum/iter;		
}

__device__ void  injector1(unsigned int arrayLoc,unsigned int posLoc,float time,float4* pos, float* pdata){
    float rnd1,rnd2,rnd3;
    rnd1 = distRnd1(pdata[arrayLoc+4] , 5);
    rnd2 = distRnd1(pdata[arrayLoc+5] , 5);
    rnd3 = distRnd1(pdata[arrayLoc+6] , 5);


    pdata[arrayLoc+1] = 0.02 * (sin(time/5  + (rnd1)/50) * (rnd2 +1)) ;//x vloocity 
    pdata[arrayLoc+2] = 0.002 * rnd3;
    pdata[arrayLoc+3] = 0.02 * (cos(time/5  + (rnd1)/50)  *(rnd2 +1));	//y volocity

    pos[posLoc].x = 0;
    //pos[posLoc].y = pdata[7];
    pos[posLoc].y = 0;
    pos[posLoc].z = 0;

}
__device__ void  injector2(unsigned int arrayLoc,unsigned int posLoc,int injNum,float time,float3* pos, float* pdata,float* debugData){
    float rnd1,rnd2,rnd3,rnd4,rnd5;
    float dt,du,dx,dy,dz,dx2,dy2,dz2,len,vx,vy,vz,dxt,dyt,dzt,dxu,dyu,dzu;
    // float dv
    /*
       injdata[injNum][1][0]// type, injection ratio ie streem volume, ~
       injdata[injNum][2][0];//x,y,z position
       injdata[injNum][3][0];//x,y,z velocity
       injdata[injNum][4][0];//x,y,z size
       injdata[injNum][5][0];//t,u,v jiter v not implemented = speed 
       injdata[injNum][6][0];//speed jiter ~ 
       injdata[injNum][7][0];//centrality of rnd distribution speed dt tu 

     */
    //if ((pdata[arrayLoc+4] +1) /2 <  injdata[injNum][1][1]){ return;}// reterns without injection ?????


    rnd1 = (distRnd1(pdata[arrayLoc+4] , (int)injdata[injNum][7][0])+1)/2;
    rnd2 = (distRnd1(pdata[arrayLoc+5] , (int)injdata[injNum][7][1])+1)/2;
    rnd3 = (distRnd1(pdata[arrayLoc+6] , (int)injdata[injNum][7][2])+1)/2;
    rnd4 = (distRnd1(pdata[arrayLoc+4],1) );
    rnd5 = (distRnd1(pdata[arrayLoc+5],1) );
    //float rnd6 = (distRnd1(pdata[arrayLoc+6],1) );
    vx = injdata[injNum][3][0];vy = injdata[injNum][3][1];vz = injdata[injNum][3][2];//direction of spray	

    dt = injdata[injNum][5][0];du = injdata[injNum][5][1];// dv = injdata[injecti +17] * 0;// z component not implemented jitterelitive to direction of spreay



    // vector vx,vy,vz X 0,1,0
    dx = -vz;dy = 0;dz = vx;//  dt directon

    len = sqrt(dx*dx +dy*dy + dz*dz);
    if (len ==0)
    {
	dx = 0;dy =0;dz =0;
    }
    else{
	dx =dx/len;dy =dy/len;dz =dz/len;
    }
    //scale by dt amout of jitter in dt direction
    dxt = dx *dt;dyt = dy * dt;dzt = dz *dt;

    // vector vx,vy,vz X 0,1,0 X vx,vy,vz 
    dx2 = vy*vx;dy2 = vz*vz-vx*vx;dz2 = vy*vz;// du direction
    len = sqrt(dx2*dx2 +dy2*dy2 + dz2*dz2);
    if (len ==0)
    {
	dx2 = 0;dy2 =0;dz2 =0;
    }
    else{
	dx2 =dx2/len;dy2 =dy2/len;dz2 =dz2/len;
    }
    //scale by du amout of jutter in du direction
    dxu = dx2 *du;dyu = dy2 * du;dzu = dz2 *du;


    //indesices num injectors =0,position =6,velosity =9, size =12 tuv jiter = 15,speed = 18,centrality of randum  
    //         3 +             speed component                          velocity          t jitter u jitter
    if (injdata[injNum][1][0] ==1)
    {
	pdata[arrayLoc+1] = ( rnd1 * injdata[injNum][6][0]) * (injdata[injNum][3][0] + dxt * rnd2 + dxu * rnd3) ;//x vloocity 
	pdata[arrayLoc+2] = ( rnd1  * injdata[injNum][6][0]) * (injdata[injNum][3][1] + dyt * rnd2+ dyu * rnd3) ; // y velocity
	pdata[arrayLoc+3] = ( rnd1  * injdata[injNum][6][0]) * (injdata[injNum][3][2] + dzt * rnd2+ dzu * rnd3);	//z volocity
    }
    if (injdata[injNum][1][0] ==2)
    {
	pdata[arrayLoc+1] = ( rnd1 * injdata[injNum][6][1]+ injdata[injNum][6][0]) * (injdata[injNum][3][0] + dxt * rnd2 + dxu * rnd3) ;//x vloocity 
	pdata[arrayLoc+2] = ( rnd1  * injdata[injNum][6][1] + injdata[injNum][6][0]) * (injdata[injNum][3][1] + dyt * rnd2+ dyu * rnd3) ; // y velocity
	pdata[arrayLoc+3] = ( rnd1  * injdata[injNum][6][1] +injdata[injNum][6][0]) * (injdata[injNum][3][2] + dzt * rnd2+ dzu * rnd3);	//z volocity
    }
    // size computation  xform  to dt du dv

    dt = injdata[injNum][4][0];du = injdata[injNum][4][1];//dv = injdata[injecti +14] * 0;//re use varables z component not implemented jitterelitive to direction of spreay
    dxt = dx *dt;dyt = dy * dt;dzt = dz *dt;
    dxu = dx2 *du;dyu = dy2 * du;dzu = dz2 *du;

    if (injdata[injNum][1][0] ==1)
    {
	pos[posLoc].x = injdata[injNum][2][0] +  dxt * rnd4 + dxu * rnd5;   
	pos[posLoc].y = injdata[injNum][2][1] + dyt * rnd4 + dyu * rnd5 ; 
	pos[posLoc].z = injdata[injNum][2][2]  + dzt * rnd4+ dzu * rnd5;
    }


    if (injdata[injNum][1][0] ==2)
    {
	pos[posLoc].x = injdata[injNum][2][0] +  injdata[injNum][4][0] * distRnd1(pdata[arrayLoc+4] , 3);   
	pos[posLoc].y = injdata[injNum][2][1] +  injdata[injNum][4][1] * distRnd1(pdata[arrayLoc+5] , 3); 
	pos[posLoc].z = injdata[injNum][2][2]  +  injdata[injNum][4][2] * distRnd1(pdata[arrayLoc+6] , 3);
    }




    if (DEBUG == 1)
    {

	int dbi =0;
	debugData[dbi + 0] = (float)injNum ;debugData[dbi + 1] =  injdata[injNum][1][1];debugData[dbi + 2] =0;
	dbi=dbi+3;
	debugData[dbi + 0] = dx;debugData[dbi + 1] = dy;debugData[dbi + 2] = dz;
	dbi=dbi+3;
	debugData[dbi + 0] = dx2;debugData[dbi + 1] = dy2;debugData[dbi + 2] = dz2;
	dbi=dbi+3;
	debugData[dbi + 0] = dxt;debugData[dbi + 1] = dyt;debugData[dbi + 2] = dzt;
	dbi=dbi+3;
	debugData[dbi + 0] = dxu;debugData[dbi + 1] = dyu;debugData[dbi + 2] = dzu;

    }

}
///////////////////////////////////////////////////////////////////////
__device__ void  planeReflector1(float posX,float posY,float posZ,unsigned int arrayLoc,unsigned int posLoc,int reflNum,float time,float3* pos, float* pdata,float* debugData)
{
    float xn =1,yn =1,zn =0, rad =1,damping =.7,noTraping;
    float xp,yp,zp;

    //indexices num injectors =0,position =[reflNum][1][0],normal =[reflNum][2][0], size =[reflNum][3][0] tuv jiter = [reflNum ][4][0],damping = [reflNum ][4][0],centrality of randum = 21

    //dataorginization  refldata[reflNum][rownum][quardinare numbr x=0,1=y,2=z]
    //dataorginization  type rownum 0 ~~ ,position 1,normal 2,radis 3,reflection coef 5,

    xn = refldata[reflNum][2][0];yn = refldata[reflNum][2][1];zn = refldata[reflNum][2][2];//normal
    rad = refldata[reflNum][3][0];
    damping = refldata[reflNum][5][0];
    noTraping = refldata[reflNum][5][1];
    xp = refldata[reflNum][1][0];yp = refldata[reflNum][1][1];zp = refldata[reflNum][1][2];//reflector position



    float length = sqrt(xn * xn + yn * yn + zn * zn);
    xn = xn/length;
    yn = yn/length;
    zn = zn/length;

    float distx = posX - xp;//point position - reflector position
    float disty = posY - yp;
    float distz = posZ - zp;


    float xv = pdata[arrayLoc+1];float yv = pdata[arrayLoc+2];float zv = pdata[arrayLoc+3];

    //	   	if ((fabs(distx) <= rad) && (fabs(disty)<= rad) && (fabs(distz) <= rad))
    if ((distx * distx + disty * disty + distz * distz) <= rad * rad)

    {

	if ((distx * xn + disty * yn + distz * zn) <=0)

	{
	    if ((REFL_HITS == 1) && (noTraping ==1))
	    {

		if(reflNum < 128) debugData[reflNum] = debugData[reflNum] +1;		
	    }

	    float ndotv = xv * xn + yv * yn + zv * zn;

	    float newVX =(xv - 2.0*ndotv*xn);
	    float newVY =(yv - 2.0*ndotv*yn);
	    float newVZ =(zv - 2.0*ndotv*zn);
	    // experments to lower traping  did not work
	    //damping =1;
	    //one iteration wothout damping to prevent capture.
	    pos[posLoc].x  = posX + noTraping * newVX;
	    pos[posLoc].y  = posY + noTraping * newVY;
	    pos[posLoc].z  = posZ + noTraping * newVZ;

	    pdata[arrayLoc+1] = newVX*damping;
	    pdata[arrayLoc+2] = newVY*damping;
	    pdata[arrayLoc+3] = newVZ*damping;

	    //pdata[arrayLoc] = 0;// temp set age to 0
	    if ((noTraping ==1)&& (refldata[reflNum][0][1]) == 1 )
	    {
		//pdata[arrayLoc] = pdata[arrayLoc]/2.0;// chnage color

		pdata[arrayLoc] = pdata[arrayLoc] * 1.5;// chnage color
	    }
	}
    }


}

__global__ void Point1(float3* pos, float4* color, float * pdata,float * debugData ,unsigned int width,
	unsigned int height, int max_age,int disappear_age,float alphaControl, float time, float gravity, float colorFreq, float r3)
{

    // r1,r2,r3 curently not used
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned int arrayLoc = y*width*PDATA_ROW_SIZE + x*PDATA_ROW_SIZE;
    unsigned int posLoc = y*width+x;
    float newX,newY,newZ,posX,posY,posZ;

    /*
       arrayLoc is data index of partical in pdata
       pdata [arrayLoc] = age ,pdata[arrayLoc +1 +2  +3 = x ,y,z velocity +rd1 rnd2 rnd3

       posLoc is index of partical location and[width*height + posLoc] index of color
       pos[posLoc].x .y .z  is x,y,z pos
       pos[ [width*height + posLoc].x .y .z is red green blue color
       In lines mode vbo is twice as big with tail and head psitions
     */


    if (pdata[arrayLoc] >= max_age)
    {

	int injecNum = ((arrayLoc/PDATA_ROW_SIZE) % (int) injdata[0][0][0]) +1;// pdata row mod number of injectors 
	if(( injdata[injecNum][1][1]) )  injector2(arrayLoc,posLoc,injecNum,time,pos,pdata,debugData);
	pdata[arrayLoc] = 0;//set age to 0

    }

    posX=pos[posLoc].x;posY=pos[posLoc].y;posZ=pos[posLoc].z;




    // reflector

    for (int reflNum = 1;reflNum <= refldata[0][0][0]  ;reflNum ++)
    {

	//planeReflector1( pos[posLoc].x, pos[posLoc].y, pos[posLoc].z,arrayLoc,posLoc,reflNum,time,pos,pdata,debugData);			
	if (refldata[reflNum][0][0] ==1)planeReflector1(posX,posY,posZ,arrayLoc,posLoc,reflNum,time,pos,pdata,debugData);			
    }

    pdata[arrayLoc] += 1;        // increase age
    pdata[arrayLoc+2] -= gravity; // gravity

    { // add velocity to position  ie intigrate but not store in pos[]
	posX=pos[posLoc].x;posY=pos[posLoc].y;posZ=pos[posLoc].z;// plane reflector modifyes position info
	newX = posX + pdata[arrayLoc+1];
	newY = posY + pdata[arrayLoc+2];
	newZ = posZ + pdata[arrayLoc+3];
    }



    {
	color[posLoc].y = (cos(colorFreq * 2.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;//green
	color[posLoc].x = (cos(colorFreq * 1.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;//red
	color[posLoc].z = (cos(colorFreq * 4.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;//blue
	float alpha =1; 
	if ((alphaControl == 1) && (newY <=.1)) alpha =0; 

	color[posLoc].w = alpha;//alpha
	// write output vertex
	if (pdata[arrayLoc] > disappear_age){pdata[arrayLoc+1] =10000;pdata[arrayLoc+2] =10000;pdata[arrayLoc+3] =10000;}

	pos[posLoc] = make_float3(newX, newY, newZ);
    }

}

__global__ void PointSquars(float4* pos, float * pdata, unsigned int width,
	unsigned int height, int max_age, float time, float r1, float r2, float r3)
{

    // r1,r2,r3 curently not used
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned int arrayLoc = y*width*4 + x*4;
    unsigned int posLoc = y*width+x;
    float newX,newY,newZ;

    /*
       arrayLoc is data index of partical in pdata
       pdata [arrayLoc] = age ,pdata[arrayLoc +1 +2  +3 = x ,y,z velocity

       posLoc is index of partical location and[width*height + posLoc] index of color
       pos[posLoc].x .y .z  is x,y,z pos
       pos[ [width*height + posLoc].x .y .z is red green blue color
       In lines mode vbo is twice as big with tail and head psitions
     */

    if (pdata[arrayLoc] >= max_age)
    {
	pdata[arrayLoc] = 0;

	pdata[arrayLoc+1] = 0.002 * (sin(time) + (float)x / (float)width/10.0f ) ;//x velocity  sin(time) + x index/width, x is randomly selected because of randon nature of age


	pdata[arrayLoc+2] = 0;

	pdata[arrayLoc+3] = 0.002 * (cos(time) + (float)(y) / (float)( height)/10.0f );// y velocity



	// maybe move the generation point around?


	{
	    pos[posLoc].x = 0;
	    pos[posLoc].y = 0.5;
	    pos[posLoc].z = 0;
	}

    }

    // add velocity to position  ie intigrate
    {
	newX = pos[posLoc].x + pdata[arrayLoc+1];
	newY = pos[posLoc].y + pdata[arrayLoc+2];
	newZ = pos[posLoc].z + pdata[arrayLoc+3];
    }


    pdata[arrayLoc] += 1;        // increase age
    pdata[arrayLoc+2] -= 10.1; // gravity

    // check aganst tabletop surface reverse velocity
    {

	if ((newY <= 0) && fabs(pos[posLoc].x)<5 && fabs(pos[posLoc].z)<5)
	{
	    //pdata[arrayLoc+2] = -0.7 * pdata[arrayLoc+2];
	}
    }



    // now need to modify the color info in the array
    //      pos[width*height + posLoc].x = 0.0f;//red
    //      pos[width*height + posLoc].y = 1.0f;//green
    //      pos[width*height + posLoc].z = 0.0f;//blue
    float colorFreq = 16.0f;

    {
	pos[width*height + posLoc].y = (cos(colorFreq * 2.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;
	pos[width*height + posLoc].x = (cos(colorFreq * 1.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;
	pos[width*height + posLoc].z = (cos(colorFreq * 4.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;
	// write output vertex
	pos[posLoc] = make_float4(newX, newY, newZ, 1.0f);
    }


}





